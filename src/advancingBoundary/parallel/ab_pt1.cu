#include "hip/hip_runtime.h"
/*
 * ABserial.c
 *
 *  Created on: Nov 11, 2014
 *      Author: nathan
 */

#include "ab_pt1.h"

extern "C" {
#include "boundBox.h"
}

extern "C" {
#include "computeAuxiliaryGrid_pt1.h"
}

extern "C" {
#include "compactAuxiliaryGrid_pt1.h"
}
//#include "writecell.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/resource.h>
#include <sys/times.h>
#include <sys/time.h>
#include "cuda_utils.h"
#include "timer.h"
#include <time.h>

__global__
void wd_ab_parallel_t1(double *xc, double *yc, double *xf, double *yf, double *xbox, double *ybox, struct cell_pt1 *compAuxCells, int size_c, int size_f, int numAuxCells, double auxDiag, double *wallDist){


	int myId, includesAuxCells, j, index;
	double r, rtemp, rAux;

	myId = threadIdx.x + blockDim.x * blockIdx.x;

	// Keep array access bounded
	if (myId >= size_c){
		return;
	}

	// Compute initial radius
	r=1e9;
	for (j=0; j<8; j++){
		rtemp = sqrt( pow((xc[myId]-xbox[j]),2) + pow((yc[myId]-ybox[j]),2) );
		if (rtemp<r){
			r=rtemp;
		}
	}



	// Loop through compacted auxCell array to see if any lie within rc
	includesAuxCells = 0;
	while(includesAuxCells == 0){
		for (j=0; j<numAuxCells; j++){

			rAux = sqrt( pow(xc[myId]-compAuxCells[j].xcenter,2) + pow(yc[myId]-compAuxCells[j].ycenter,2) );
			// Increase r to be sure enough geometry is included
			if(rAux < r){
				r += auxDiag*0.5;
				includesAuxCells=1;
				break;
			}
			else{
				r += auxDiag;
			}
//			if(myId==0){
//				printf("rAux, r: %f, %f\n",rAux,r);
//			}
		}

	}


	/*
	 *  Loop through compacted auxCell array. For those that lie within r,
	 *  traverse through faces, compute wallDist and check for minimum
	 */
	for (j=0; j<numAuxCells; j++){

		rAux = sqrt( pow(xc[myId]-compAuxCells[j].xcenter,2) + pow(yc[myId]-compAuxCells[j].ycenter,2));

		// Check if auxCell is within radius of interest
		if(rAux < r){
			index = 0;

			// Loop through faces and compute distance from grid cell center
			while(index < compAuxCells[j].numFaces){
				rtemp = sqrt( pow(xc[myId]-xf[compAuxCells[j].faceIndex[index]],2) + pow(yc[myId]-yf[compAuxCells[j].faceIndex[index]],2));

				// If dist is smaller than current wallDist, replace
				if(rtemp<wallDist[myId]){
					wallDist[myId]=rtemp;
				}

				index++;
			}
		}

	}





}


////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////////////


__global__
void wd_ab_parallel_t2(double *cellCenters, double *xf, double *yf, double *xbox, double *ybox, struct cell_pt1 *compAuxCells, int size_c, int size_f, int numAuxCells, double auxDiag, double *wallDist){

	extern __shared__ double s_cellCenters[];


	int myId, tid, includesAuxCells, j, index;
	double r, rtemp, rAux;

	tid = threadIdx.x;
	myId = threadIdx.x + blockDim.x * blockIdx.x;


	// Pull cell centers into shared memory
	s_cellCenters[2*tid] = cellCenters[2*myId];
	s_cellCenters[2*tid+1] = cellCenters[2*myId+1];



	// Keep array access bounded
	if (myId >= size_c){
		return;
	}

	// Compute initial radius
	r=1e9;
	for (j=0; j<8; j++){
		rtemp = sqrt( pow((s_cellCenters[2*tid]-xbox[j]),2) + pow((s_cellCenters[2*tid+1]-ybox[j]),2) );
		if (rtemp<r){
			r=rtemp;
		}
	}



	// Loop through compacted auxCell array to see if any lie within rc
	includesAuxCells = 0;
	while(includesAuxCells == 0){
		for (j=0; j<numAuxCells; j++){

			rAux = sqrt( pow(s_cellCenters[2*tid]-compAuxCells[j].xcenter,2) + pow(s_cellCenters[2*tid+1]-compAuxCells[j].ycenter,2) );
			// Increase r to be sure enough geometry is included
			if(rAux < r){
				r += auxDiag*0.5;
				includesAuxCells=1;
				break;
			}
			else{
				r += auxDiag;
			}
//			if(myId==0){
//				printf("rAux, r: %f, %f\n",rAux,r);
//			}
		}

	}


	/*
	 *  Loop through compacted auxCell array. For those that lie within r,
	 *  traverse through faces, compute wallDist and check for minimum
	 */
	for (j=0; j<numAuxCells; j++){

		rAux = sqrt( pow(s_cellCenters[2*tid]-compAuxCells[j].xcenter,2) + pow(s_cellCenters[2*tid+1]-compAuxCells[j].ycenter,2));

		// Check if auxCell is within radius of interest
		if(rAux < r){
			index = 0;

			// Loop through faces and compute distance from grid cell center
			while(index < compAuxCells[j].numFaces){
				rtemp = sqrt( pow(s_cellCenters[2*tid]-xf[compAuxCells[j].faceIndex[index]],2) + pow(s_cellCenters[2*tid+1]-yf[compAuxCells[j].faceIndex[index]],2));

				// If dist is smaller than current wallDist, replace
				if(rtemp<wallDist[myId]){
					wallDist[myId]=rtemp;
				}

				index++;
			}
		}

	}





}


//////////////////////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////



__global__
void wd_ab_parallel_t3(double *cellCenters, double *faceCenters, double *xbox, double *ybox, struct cell_pt1 *compAuxCells, int size_c, int size_f, int numAuxCells, double auxDiag, double *wallDist){

	extern __shared__ double s_faceCenters[];


	int myId, tid, includesAuxCells, i, j, index;
	double r, rtemp, rcurrent, rAux;

	tid = threadIdx.x;
	myId = threadIdx.x + blockDim.x * blockIdx.x;


	// Pull face centers into shared memory
	if (tid==0){
		for (i=0; i<size_f; i++){
			s_faceCenters[2*i] = faceCenters[2*i];
			s_faceCenters[2*i+1] = faceCenters[2*i+1];
		}
	}
	__syncthreads();

	// Keep array access bounded
	if (myId >= size_c){
		return;
	}

	// Compute initial radius
	r=1e9;
	for (j=0; j<8; j++){
		rtemp = sqrt( pow((cellCenters[2*myId]-xbox[j]),2) + pow((cellCenters[2*myId+1]-ybox[j]),2) );
		if (rtemp<r){
			r=rtemp;
		}
	}



	// Loop through compacted auxCell array to see if any lie within rc
	includesAuxCells = 0;
	while(includesAuxCells == 0){
		for (j=0; j<numAuxCells; j++){

			rAux = sqrt( pow(cellCenters[2*myId]-compAuxCells[j].xcenter,2) + pow(cellCenters[2*myId+1]-compAuxCells[j].ycenter,2) );
			// Increase r to be sure enough geometry is included
			if(rAux < r){
				r += auxDiag*0.5;
				includesAuxCells=1;
				break;
			}
			else{
				r += auxDiag;
			}
//			if(myId==0){
//				printf("rAux, r: %f, %f\n",rAux,r);
//			}
		}

	}


	/*
	 *  Loop through compacted auxCell array. For those that lie within r,
	 *  traverse through faces, compute wallDist and check for minimum
	 */
	for (j=0; j<numAuxCells; j++){

		rAux = sqrt( pow(cellCenters[2*myId]-compAuxCells[j].xcenter,2) + pow(cellCenters[2*myId+1]-compAuxCells[j].ycenter,2));

		// Check if auxCell is within radius of interest
		if(rAux < r){
			index = 0;

			// Loop through faces and compute distance from grid cell center
			while(index < compAuxCells[j].numFaces){
				rtemp = sqrt( pow(cellCenters[2*myId]-s_faceCenters[compAuxCells[j].faceIndex[index]],2) + pow(cellCenters[2*myId+1]-s_faceCenters[compAuxCells[j].faceIndex[index]],2));

				// If dist is smaller than current wallDist, replace
				if(rtemp<rcurrent){
//					wallDist[myId]=rtemp;
					rcurrent = rtemp;

				}

				index++;
			}
		}

	}

	// Store wallDistance to global array
	wallDist[myId] = rcurrent;


}





//////////////////////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////


__global__
void wd_ab_parallel_t4(double *cellCenters, double *faceCenters, double *xbox, double *ybox, struct cell_pt1 *compAuxCells, int size_c, int size_f, int numAuxCells, double auxDiag, double *wallDist){



	int myId, tid, includesAuxCells, i, j, index;
	double r, rtemp, rcurrent, rAux, c_x, c_y;

	tid = threadIdx.x;
	myId = threadIdx.x + blockDim.x * blockIdx.x;


	// Keep array access bounded
	if (myId >= size_c){
		return;
	}


	c_x = cellCenters[2*myId];
	c_y = cellCenters[2*myId+1];



	// Compute initial radius
	r=1e9;
	for (j=0; j<8; j++){
		rtemp = sqrt( pow((c_x-xbox[j]),2) + pow((c_y-ybox[j]),2) );
		if (rtemp<r){
			r=rtemp;
		}
	}



	// Loop through compacted auxCell array to see if any lie within rc
	includesAuxCells = 0;
	while(includesAuxCells == 0){
		for (j=0; j<numAuxCells; j++){

			rAux = sqrt( pow(c_x-compAuxCells[j].xcenter,2) + pow(c_y-compAuxCells[j].ycenter,2) );
			// Increase r to be sure enough geometry is included
			if(rAux < r){
				r += auxDiag*0.5;
				includesAuxCells=1;
				break;
			}
			else{
				r += auxDiag;
			}

		}

	}


	/*
	 *  Loop through compacted auxCell array. For those that lie within r,
	 *  traverse through faces, compute wallDist and check for minimum
	 */
	for (j=0; j<numAuxCells; j++){

		rAux = sqrt( pow(c_x-compAuxCells[j].xcenter,2) + pow(c_y-compAuxCells[j].ycenter,2));

		// Check if auxCell is within radius of interest
		if(rAux < r){
			index = 0;

			// Loop through faces and compute distance from grid cell center
			while(index < compAuxCells[j].numFaces){
				rtemp = sqrt( pow(c_x-faceCenters[compAuxCells[j].faceIndex[index]],2) + pow(c_y-faceCenters[compAuxCells[j].faceIndex[index]],2));

				// If dist is smaller than current wallDist, replace
				if(rtemp<rcurrent){
//					wallDist[myId]=rtemp;
					rcurrent = rtemp;

				}

				index++;
			}
		}

	}

	// Store wallDistance to global array
	wallDist[myId] = rcurrent;


}





//////////////////////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////


__global__
void wd_ab_parallel_t5(double *cellCenters, double *faceCenters, double *xbox, double *ybox, struct cell_pt1 *compAuxCells, int size_c, int size_f, int numAuxCells, double auxDiag, double *wallDist){

	extern __shared__ double s_faceCenters [];


	int myId, tid, includesAuxCells, i, j, index, f_start, f_end, s_ind;
	double r, rtemp, rcurrent, rAux, c_x, c_y;

	tid = threadIdx.x;
	myId = threadIdx.x + blockDim.x * blockIdx.x;


	// Keep array access bounded
	if (myId >= size_c){
		return;
	}

	f_start=0;
	f_end=1024;


	// Loop through shared memory cycles until all faces have been processed
//	while


		// Pull 1024 faces into shared memory
		if (tid==0){
			s_ind = 0;
			for (i=f_start; i<f_end; i++){
				s_faceCenters[2*i] = faceCenters[2*myId];
				s_faceCenters[2*i+1] = faceCenters[2*myId+1];
			}
		}

	c_x = cellCenters[2*myId];
	c_y = cellCenters[2*myId+1];



	// Compute initial radius
	r=1e9;
	for (j=0; j<8; j++){
		rtemp = sqrt( pow((c_x-xbox[j]),2) + pow((c_y-ybox[j]),2) );
		if (rtemp<r){
			r=rtemp;
		}
	}



	// Loop through compacted auxCell array to see if any lie within rc
	includesAuxCells = 0;
	while(includesAuxCells == 0){
		for (j=0; j<numAuxCells; j++){

			rAux = sqrt( pow(c_x-compAuxCells[j].xcenter,2) + pow(c_y-compAuxCells[j].ycenter,2) );
			// Increase r to be sure enough geometry is included
			if(rAux < r){
				r += auxDiag*0.5;
				includesAuxCells=1;
				break;
			}
			else{
				r += auxDiag;
			}

		}

	}


	/*
	 *  Loop through compacted auxCell array. For those that lie within r,
	 *  traverse through faces, compute wallDist and check for minimum
	 */
	for (j=0; j<numAuxCells; j++){

		rAux = sqrt( pow(c_x-compAuxCells[j].xcenter,2) + pow(c_y-compAuxCells[j].ycenter,2));

		// Check if auxCell is within radius of interest
		if(rAux < r){
			index = 0;

			// Loop through faces and compute distance from grid cell center
			while(index < compAuxCells[j].numFaces){
				rtemp = sqrt( pow(c_x-faceCenters[compAuxCells[j].faceIndex[index]],2) + pow(c_y-faceCenters[compAuxCells[j].faceIndex[index]],2));

				// If dist is smaller than current wallDist, replace
				if(rtemp<rcurrent){
//					wallDist[myId]=rtemp;
					rcurrent = rtemp;

				}

				index++;
			}
		}

	}

	// Store wallDistance to global array
	wallDist[myId] = rcurrent;


}





//////////////////////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////



void ab_parallel_t1(double * xc, double * yc, double * xf, double * yf, int size_c, int size_f, double * wallDist){

	double xmin;
	double xmax;
	double ymin;
	double ymax;

	////////////////////////////////////////////////////////////////////
	//		Pre-processing
	////////////////////////////////////////////////////////////////////

	// Create geometry bounding box
	boundBox(xf,yf,size_f,&xmin,&xmax,&ymin,&ymax);


	// Create auxiliary grid
	int resI=80;
	int resJ=80;
	double auxDiag = sqrt( pow((xmax-xmin)/(double)(resI-1),2) + pow((ymax-ymin)/(double)(resJ-1),2));


	int numAuxCells = (resI-1)*(resJ-1);
	int i, j, cellsWithFaces;
	struct cell_pt1 *auxCells;
//	auxCells = (struct cell_pt1 *)malloc(numAuxCells*sizeof(struct cell_pt1));
	auxCells = new cell_pt1[numAuxCells];


	computeAuxiliaryGrid_pt1(xmin,xmax,ymin,ymax,resI,resJ,auxCells);

	// Count number of auxiliary cells that contain geometry faces
	cellsWithFaces = 0;
	for (i=0; i<numAuxCells; i++){
		for (j=0; j<size_f; j++){

			if (xf[j] < auxCells[i].xmax && xf[j] > auxCells[i].xmin && yf[j] < auxCells[i].ymax && yf[j] > auxCells[i].ymin){
				cellsWithFaces++;
				break;
			}

		}
	}


	// Allocate memory for compacted cells
	struct cell_pt1 * compAuxCells;
//	compAuxCells = (struct cell_pt1 *)malloc(cellsWithFaces*sizeof(struct cell_pt1));
	compAuxCells = new cell_pt1[cellsWithFaces];


	///////
	compactAuxiliaryGrid_pt1(auxCells,numAuxCells,compAuxCells,xf,yf,size_f);
	///////




	// Bounding box point arrays
	double xmid = (xmax+xmin)/2.0;
	double ymid = (ymax+ymin)/2.0;

	double xBoxPts[8] = {xmin, xmid, xmax, xmax, xmax, xmid, xmin, xmin};
	double yBoxPts[8] = {ymin, ymin, ymin, ymid, ymax, ymax, ymax, ymid};



	////////////////////////////////////////////////////////////////////////////////
	//	Combine xc,yc arrays for coallesced memory access in parallel t2 version
	////////////////////////////////////////////////////////////////////////////////

	double *cellCenters;
	cellCenters = new double[2*size_c];

	for (i=0; i<size_c; i++){
		cellCenters[2*i] = xc[i];
		cellCenters[2*i+1] = yc[i];
	}


	double *faceCenters;
	faceCenters = new double[2*size_f];

	for (i=0; i<size_f; i++){
		faceCenters[2*i] = xf[i];
		faceCenters[2*i+1] = yf[i];
	}



	////////////////////////////////////////////////////////////////////
	//  Allocate device memory and copy data
	////////////////////////////////////////////////////////////////////
	// bounding box
	double *d_xbox, *d_ybox;
	checkCudaErrors(hipMalloc(&d_xbox,8*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_ybox,8*sizeof(double)));

	checkCudaErrors(hipMemcpy(d_xbox,xBoxPts,8*sizeof(double),hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_ybox,yBoxPts,8*sizeof(double),hipMemcpyHostToDevice));

	// grid and faces
	double *d_xc, *d_yc, *d_xf, *d_yf, *d_cellCenters, *d_faceCenters;
	checkCudaErrors(hipMalloc(&d_xc,size_c*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_yc,size_c*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_xf,size_c*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_yf,size_c*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_cellCenters,2*size_c*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_faceCenters,2*size_f*sizeof(double)));

	checkCudaErrors(hipMemcpy(d_xc,xc,size_c*sizeof(double),hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_yc,yc,size_c*sizeof(double),hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_xf,xf,size_c*sizeof(double),hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_yf,yf,size_c*sizeof(double),hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_cellCenters,cellCenters,2*size_c*sizeof(double),hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_faceCenters,faceCenters,2*size_f*sizeof(double),hipMemcpyHostToDevice));

	// auxCell structs
	struct cell_pt1 * d_compAuxCells;
	checkCudaErrors(hipMalloc((void **)&d_compAuxCells,cellsWithFaces*sizeof(struct cell_pt1)));
	checkCudaErrors(hipMemcpy(d_compAuxCells,compAuxCells,cellsWithFaces*sizeof(struct cell_pt1),hipMemcpyHostToDevice));


	// wallDist array
	double *d_wallDist;
	checkCudaErrors(hipMalloc(&d_wallDist,size_c*sizeof(double)));
	checkCudaErrors(hipMemcpy(d_wallDist,wallDist,size_c*sizeof(double),hipMemcpyHostToDevice));




	////////////////////////////////////////////////////////////////////
	//	Wall Distance Calc
	////////////////////////////////////////////////////////////////////
	GpuTimer timer;

	int threadsPerBlock, numBlocks;
	threadsPerBlock = 512;
	numBlocks = (size_c/threadsPerBlock)+1;



	timer.Start();
	wd_ab_parallel_t1<<<numBlocks,threadsPerBlock>>>(d_xc,d_yc,d_xf,d_yf,d_xbox,d_ybox,d_compAuxCells,size_c,size_f,cellsWithFaces,auxDiag,d_wallDist);
	timer.Stop();
	printf("Advancing boundary - parallel T1(GpuTimer): \t %.0f milliseconds\n",timer.Elapsed());


	// Reset wallDistance
	checkCudaErrors(hipMemcpy(d_wallDist,wallDist,size_c*sizeof(double),hipMemcpyHostToDevice));



	timer.Start();
	wd_ab_parallel_t2<<<numBlocks,threadsPerBlock,2*threadsPerBlock*sizeof(double)>>>(d_cellCenters,d_xf,d_yf,d_xbox,d_ybox,d_compAuxCells,size_c,size_f,cellsWithFaces,auxDiag,d_wallDist);
	timer.Stop();
	printf("Advancing boundary - parallel T2(GpuTimer): \t %.0f milliseconds\n",timer.Elapsed());


//	timer.Start();
//	wd_ab_parallel_t3<<<numBlocks,threadsPerBlock,2*size_f*sizeof(double)>>>(d_cellCenters,d_faceCenters,d_xbox,d_ybox,d_compAuxCells,size_c,size_f,cellsWithFaces,auxDiag,d_wallDist);
//	timer.Stop();
//	printf("Advancing boundary - parallel T3(GpuTimer): \t %.0f milliseconds\n",timer.Elapsed());


	timer.Start();
	wd_ab_parallel_t4<<<numBlocks,threadsPerBlock>>>(d_cellCenters,d_faceCenters,d_xbox,d_ybox,d_compAuxCells,size_c,size_f,cellsWithFaces,auxDiag,d_wallDist);
	timer.Stop();
	printf("Advancing boundary - parallel T4(GpuTimer): \t %.0f milliseconds\n",timer.Elapsed());


//	timer.Start();
//	wd_ab_parallel_t5<<<numBlocks,threadsPerBlock,2048*sizeof(double)>>>(d_cellCenters,d_faceCenters,d_xbox,d_ybox,d_compAuxCells,size_c,size_f,cellsWithFaces,auxDiag,d_wallDist);
//	timer.Stop();
//	printf("Advancing boundary - parallel T5(GpuTimer): \t %.0f milliseconds\n",timer.Elapsed());



	// Copy wallDist back to host
	checkCudaErrors(hipMemcpy(wallDist,d_wallDist,sizeof(double)*size_c,hipMemcpyDeviceToHost));







	////////////////////////////////////////////////////////////////////
	//
	////////////////////////////////////////////////////////////////////



}




